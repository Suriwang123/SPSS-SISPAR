
#include <hip/hip_runtime.h>

// ParallelSPSS.Form1
extern "C" __global__  void addVector( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N);
// ParallelSPSS.Form1
extern "C" __global__  void powerVector( float* a, int aLen0,  float* b, int bLen0, float c, int N);
// ParallelSPSS.Form1
extern "C" __global__  void multiplyVector( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N);

// ParallelSPSS.Form1
extern "C" __global__  void addVector( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x)
	{
		c[(i)] = a[(i)] + b[(i)];
	}
}
// ParallelSPSS.Form1
extern "C" __global__  void powerVector( float* a, int aLen0,  float* b, int bLen0, float c, int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x)
	{
		b[(i)] = (a[(i)] - c) * (a[(i)] - c);
	}
}
// ParallelSPSS.Form1
extern "C" __global__  void multiplyVector( float* a, int aLen0,  float* b, int bLen0,  float* c, int cLen0, int N)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += gridDim.x)
	{
		c[(i)] = a[(i)] * b[(i)];
	}
}
